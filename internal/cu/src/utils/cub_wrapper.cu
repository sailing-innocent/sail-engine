/**
 * @file cub_wrapper.cu
 * @brief The CUB Wrapper Implementation
 * @author sailing-innocent
 * @date 2024-05-05
 */

#include "SailCu/utils/cub_warpper.h"

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
// #include <iostream>

namespace sail::cu {

void cub_inclusive_sum(int* d_in, int* d_out, int N) {
	void* d_temp_storage = nullptr;
	size_t temp_storage_bytes = 0;
	hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_in, N);
	hipDeviceSynchronize();
	// std::cout << "temp_storage_bytes: " << temp_storage_bytes << std::endl;
	hipMalloc(&d_temp_storage, temp_storage_bytes);
	hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, N);
	hipFree(d_temp_storage);
}

}// namespace sail::cu
